#include "hip/hip_runtime.h"
/*
 * STRUMPACK -- STRUctured Matrices PACKage, Copyright (c) 2014, The
 * Regents of the University of California, through Lawrence Berkeley
 * National Laboratory (subject to receipt of any required approvals
 * from the U.S. Dept. of Energy).  All rights reserved.
 *
 * If you have questions about your rights to use or distribute this
 * software, please contact Berkeley Lab's Technology Transfer
 * Department at TTD@lbl.gov.
 *
 * NOTICE. This software is owned by the U.S. Department of Energy. As
 * such, the U.S. Government has been granted for itself and others
 * acting on its behalf a paid-up, nonexclusive, irrevocable,
 * worldwide license in the Software to reproduce, prepare derivative
 * works, and perform publicly and display publicly.  Beginning five
 * (5) years after the date permission to assert copyright is obtained
 * from the U.S. Department of Energy, and subject to any subsequent
 * five (5) year renewals, the U.S. Government is granted for itself
 * and others acting on its behalf a paid-up, nonexclusive,
 * irrevocable, worldwide license in the Software to reproduce,
 * prepare derivative works, distribute copies to the public, perform
 * publicly and display publicly, and to permit others to do so.
 *
 * Developers: Pieter Ghysels, Francois-Henry Rouet, Xiaoye S. Li.
 *             (Lawrence Berkeley National Lab, Computational Research
 *             Division).
 *
 */
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/reverse.h>
#include <thrust/sequence.h>
#include <thrust/logical.h>
#include <thrust/functional.h>
#include <thrust/gather.h>
#include <thrust/random/linear_congruential_engine.h>
#include <thrust/random/linear_feedback_shift_engine.h>
#include <thrust/random/uniform_int_distribution.h>

#include "ANDSparspak.hpp"

namespace strumpack {
  namespace ordering {

    template<typename integer> struct bfs_label_prop_ftor {
      integer *ptr, *ind;
      bool *eliminated, *running;
      int *label;
      bfs_label_prop_ftor(integer* ptr_, integer* ind_,
                          bool* eliminated_, int* label_,
                          bool* running_)
        : ptr(ptr_), ind(ind_), eliminated(eliminated_),
          label(label_), running(running_) {}
      __device__ void operator()(int i) {
        if (eliminated[i]) return;
        auto khi = ptr[i+1];
        for (auto k=ptr[i]; k<khi; k++) {
          auto j = ind[k];
          if (eliminated[j]) continue;
          if (label[j] < label[i]) {
            label[i] = label[j];
            // TODO atomicExch?
            *running = true;
          }
        }
      }
    };

    // #define FRONT_BFS
#if defined(FRONT_BFS)
    template<typename integer> struct bfs_layer_front_ftor {
      integer *ptr, *ind;
      bool *eliminated;
      int n, *label, *current_front, *next_front, *next_front_size;
      bfs_layer_front_ftor(int n_, integer* ptr_, integer* ind_,
                           bool* eliminated_, int* label_,
                           int* current_front_, int* next_front_,
                           int* next_front_size_)
        : n(n_), ptr(ptr_), ind(ind_), eliminated(eliminated_),
          label(label_), current_front(current_front_),
          next_front(next_front_), next_front_size(next_front_size_) {}
      __device__ void operator()(int fi) {
        auto i = current_front[fi];
        auto li1 = label[i]+1;
        auto khi = ptr[i+1];
        for (auto k=ptr[i]; k<khi; k++) {
          auto j = ind[k];
          if (eliminated[j]) continue;
          if (atomicCAS(label+j, n, li1) == n)
            next_front[atomicAdd(next_front_size, 1)] = j;
        }
      }
    };
#else
    template<typename integer> struct bfs_layer_ftor {
      integer *ptr, *ind;
      bool *eliminated, *running;
      int *label, *comp, *comp_start, n, it;
      bfs_layer_ftor(int n_, int it_, integer* ptr_, integer* ind_,
                     bool* eliminated_, int* label_, int* comp_,
                     int* comp_start_, bool* running_)
        : n(n_), it(it_), ptr(ptr_), ind(ind_), eliminated(eliminated_),
          label(label_), comp(comp_), comp_start(comp_start_),
          running(running_) {}
      __device__ void operator()(int i) {
        // TODO write from the destination instead of the source?
        if (eliminated[i]) return;
        auto li = label[i];
        if (li == comp_start[comp[i]] + it) {
          bool run = false;
          auto khi = ptr[i+1];
          for (auto k=ptr[i]; k<khi; k++) {
            auto j = ind[k];
            if (eliminated[j]) continue;
            if (atomicCAS(label+j, n, li+1) == n)
              run = true;
          }
          if (run && !*running)
            // TODO atomicExch ?
            *running = true;
        }
      }
    };
#endif

    struct find_comps_ftor {
      int *nr_comps, *label, *comp, *comp_root;
      bool *eliminated;
      find_comps_ftor(int* nr_comps_, bool* eliminated_, int* label_,
                      int* comp_, int* comp_root_)
        : nr_comps(nr_comps_), eliminated(eliminated_), label(label_),
          comp(comp_), comp_root(comp_root_) {}
      __device__ void operator()(int i) {
        if (!eliminated[i] && label[i] == i) {
          auto c = atomicAdd(nr_comps, 1);
          comp[i] = c;
          comp_root[c] = i;
        }
      }
    };

    struct count_comps_ftor {
      bool *eliminated;
      int nr_comps, *comp, *comp_size;
      count_comps_ftor(bool* eliminated_, int nr_comps_,
                       int* comp_, int* comp_size_)
        : eliminated(eliminated_), nr_comps(nr_comps_), comp(comp_),
          comp_size(comp_size_) {}
      __device__ void operator()(int i) {
        if (!eliminated[i])
          atomicAdd(comp_size+comp[i], 1);
        else
          comp[i] = nr_comps;
      }
    };

    struct random_roots_ftor {
      int seed, *root, *start, *map;
      random_roots_ftor(int seed_, int* root_, int* start_, int* map_)
        : seed(seed_), root(root_), start(start_), map(map_) {}
      __device__ //__forceinline__
      void operator()(int i) {
        thrust::minstd_rand gen(seed);
        thrust::uniform_int_distribution<int>
          dist(start[i], start[i+1]-1);
        root[i] = map[dist(gen)];
      }
    };

    struct count_layers_ftor {
      bool *eliminated;
      int n, *label, *comp, *levels, *comp_start;
      count_layers_ftor(int n_, bool* eliminated_, int* label_,
                        int* comp_, int* levels_, int* comp_start_)
        : n(n_), eliminated(eliminated_), label(label_), comp(comp_),
          levels(levels_), comp_start(comp_start_) {}
      __device__ void operator()(int i) {
        if (eliminated[i]) return;
        auto li = label[i];
        if (li == n) return;
        auto ci = comp[i];
        auto lvl = li - comp_start[ci] + 1;
        if (lvl > levels[ci])
          atomicMax(levels+ci, lvl);
      }
    };

    template<typename integer> struct peripheral_node_1_ftor {
      integer *ptr, *ind;
      bool *eliminated;
      int *label, *comp, *levels, *old_levels, *d, *comp_start;
      peripheral_node_1_ftor(integer* ptr_, integer* ind_,
                             bool* eliminated_, int* label_, int* comp_,
                             int* levels_, int* old_levels_,
                             int* d_, int* comp_start_)
        : ptr(ptr_), ind(ind_), eliminated(eliminated_),
          label(label_), comp(comp_),
          levels(levels_), old_levels(old_levels_),
          d(d_), comp_start(comp_start_) {}
      __device__ void operator()(int i) {
        // find the node with the minimum degree in the last layer, for
        // each of the components. We cannot atomically update both the
        // degree and the index of the node which had the minimum
        // degree, hence the two loops.
        if (eliminated[i]) return;
        auto ci = comp[i];
        auto nlvls = levels[ci];
        if (nlvls <= old_levels[ci]) return;
        if (label[i] == comp_start[ci] + nlvls - 1) {
          // in the last layer
          int deg = 0;
          auto khi = ptr[i+1];
          for (auto k=ptr[i]; k<khi; k++)
            if (!eliminated[ind[k]])
              deg++;
          atomicMin(d+ci, deg);
        }
      }
    };

    template<typename integer> struct peripheral_node_2_ftor {
      integer *ptr, *ind;
      bool *eliminated;
      int *label, *comp, *levels, *old_levels,
        *d, *comp_start, *comp_root;
      peripheral_node_2_ftor(integer* ptr_, integer* ind_,
                             bool* eliminated_, int* label_, int* comp_,
                             int* levels_, int* old_levels_, int* d_,
                             int* comp_start_, int* comp_root_)
        : ptr(ptr_), ind(ind_), eliminated(eliminated_), label(label_),
          comp(comp_), levels(levels_), old_levels(old_levels_), d(d_),
          comp_start(comp_start_), comp_root(comp_root_) {}
      __device__ void operator()(int i) {
        if (eliminated[i]) return;
        auto ci = comp[i];
        auto nlvls = levels[ci];
        if (nlvls <= old_levels[ci]) return;
        if (label[i] == comp_start[ci] + nlvls - 1) {
          // in the last layer
          int deg = 0;
          auto khi = ptr[i+1];
          for (auto k=ptr[i]; k<khi; k++)
            if (!eliminated[ind[k]])
              deg++;
          if (deg == d[ci])
            // multiple threads might be writing concurrently here if
            // multiple nodes have the same (minimum) degree
            //atomicExch(comp_root+ci, i);
            comp_root[ci] = i;
        }
      }
    };

    // Store layer cardinalities for the component starting at node i
    // (== label[i]) at layer_size[i], and at layer_size[i+1] for the
    // next layer, etc. Count nr of nodes in each separator, where a
    // separator is all nodes in a layer that are also connected to
    // the next layer, store result in sep_size[l] for level l.
    template<typename integer> struct cardinalities_ftor {
      integer *ptr, *ind;
      bool *eliminated;
      int *label, *comp, *levels, *comp_start,
        *layer_size, *sep_size;
      cardinalities_ftor(integer* ptr_, integer* ind_,
                         bool* eliminated_, int* label_, int* comp_,
                         int* levels_, int* comp_start_, int* layer_size_,
                         int* sep_size_)
        : ptr(ptr_), ind(ind_), eliminated(eliminated_), label(label_),
          comp(comp_), levels(levels_), comp_start(comp_start_),
          layer_size(layer_size_), sep_size(sep_size_) {}
      __device__ void operator()(int i) {
        if (eliminated[i]) return;
        auto ci = comp[i];
        auto nlvls = levels[ci];
        if (nlvls < 3) // whole component eliminated
          return;
        auto l0 = comp_start[ci];
        auto li = label[i];
        atomicAdd(layer_size+li, 1);
        if (li == l0 || li == l0+nlvls-1) // ignore first/last
          return;
        auto khi = ptr[i+1];
        for (auto k=ptr[i]; k<khi; k++) {
          auto j = ind[k];
          if (eliminated[j]) continue;
          if (label[j] == li + 1) {
            atomicAdd(sep_size+li, 1);
            return;
          }
        }
      }
    };

    template<typename integer> struct minimize_sep_ftor {
      integer *ptr, *ind;
      bool *eliminated;
      int *label, *comp, *levels, *comp_start, *layer_size,
        *sep_size, *comp_root, *comp_root_opt, *sep_level;
      float *min_norm_sep;
      minimize_sep_ftor(integer* ptr_, integer* ind_,
                        bool* eliminated_, int* label_, int* comp_,
                        int* levels_, int* comp_start_,
                        int* layer_size_, int* sep_size_,
                        int* comp_root_, int* comp_root_opt_,
                        int* sep_level_, float* min_norm_sep_)
        : ptr(ptr_), ind(ind_),
          eliminated(eliminated_), label(label_), comp(comp_),
          levels(levels_), comp_start(comp_start_),
          layer_size(layer_size_), sep_size(sep_size_),
          comp_root(comp_root_), comp_root_opt(comp_root_opt_),
          sep_level(sep_level_), min_norm_sep(min_norm_sep_) {}
      __device__ void operator()(int i) {
        if (eliminated[i]) return;
        auto ci = comp[i];
        auto nlvls = levels[ci];
        if (nlvls < 3 || label[i] != comp_start[ci]) return;
        // I'm the root of this component
        auto c0 = comp_start[ci];
        auto cA = layer_size[c0];
        auto cB = comp_start[ci+1] - c0 - cA;
        float minNS;
        int minl;
        for (auto l=1; l<nlvls-1; l++) {
          auto cL = layer_size[c0+l];
          cB -= cL;
          auto cS = sep_size[c0+l];
          auto NS = cS * (1. / (cA+cL-cS) + 1. / cB);
          if (l == 1 || NS < minNS) {
            minNS = NS;
            minl = l;
          }
          cA += cL;
        }
        if (minNS < min_norm_sep[ci]) {
          sep_level[ci] = c0 + minl;
          comp_root_opt[ci] = comp_root[ci];
          min_norm_sep[ci] = minNS;
        }
      }
    };

    template<typename integer> struct eliminate_sep_ftor {
      integer *ptr, *ind, *perm;
      bool *eliminated;
      int *label, *comp, *levels, *sep_size, *nr_eliminated;
      eliminate_sep_ftor(integer* ptr_, integer* ind_,
                         bool* eliminated_, int* label_, int* comp_,
                         int* levels_, int* sep_size_,
                         int* nr_eliminated_, integer* perm_)
        : ptr(ptr_), ind(ind_), eliminated(eliminated_), label(label_),
          comp(comp_), levels(levels_), sep_size(sep_size_),
          nr_eliminated(nr_eliminated_), perm(perm_) {}
      __device__ void operator()(int i) {
        if (eliminated[i]) return;
        auto ci = comp[i];
        if (levels[ci] < 3) {
          eliminated[i] = true;
          perm[atomicAdd(nr_eliminated, 1)] = i;
          return;
        }
        auto li = label[i];
        if (li == sep_size[ci]) { // best layer
          auto khi = ptr[i+1];
          for (auto k=ptr[i]; k<khi; k++) {
            auto j = ind[k];
            if (eliminated[j]) continue;
            if (label[j] == li + 1) {
              eliminated[i] = true;
              perm[atomicAdd(nr_eliminated, 1)] = i;
              return;
            }
          }
        }
      }
    };

    struct tuple_less_equal_ftor {
      __device__ __forceinline__
      bool operator()(const thrust::tuple<int,int>& t) {
        return thrust::get<0>(t) <= thrust::get<1>(t);
      }
    };
    struct tuple_equal_ftor {
      __device__ __forceinline__
      bool operator()(const thrust::tuple<int,int>& t) {
        return thrust::get<0>(t) == thrust::get<1>(t);
      }
    };

    template<typename integer> void
    nd_bfs_device(int n, thrust::device_vector<integer>& ptr,
                  thrust::device_vector<integer>& ind,
                  thrust::device_vector<integer>& perm) {
      thrust::device_vector<int>
        d_nr_comps(1),        // nr of connected components,
                              // stored on device
        nr_eliminated(1, 0),  // nr of already eliminated nodes
        label(n),             // node label, as used in label
                              // propagation breadth first search for
                              // finding connected components, and in
                              // the BFS to find the level sets
        comp(n),              // number of the component for each node
        degree(n),            // node degrees, used to find a new
                              // peripheral node, which is a node in
                              // the last level with the smallest
                              // degree. Also used to store the
                              // caridnalities of the levels
        levels(n),            // number of levels in the level sets
                              // (one value per component)
        old_levels(n),        // previous number of leves in the level
                              // sets (one value per component)
        comp_start(n+1),      // start index of each component (one
                              // value per component), this is an
                              // exclusive scan of the sizes of the
                              // components
        comp_root(n),         // start points for the level set BFS
                              // for each component, also used to
                              // store root of nodes
        comp_root_opt(n),     // start points for the level set BFS
                              // for each component, also used to
                              // store root of nodes
        sep_size(n),          // size of the separator corresponding
                              // to each level in the level set, where
                              // the separator is the subset of nodes
                              // in the level that are connected to
                              // the next level
        sep_level(n);         // the optimal level from the level sets
                              // to use as a separator, for each
                              // component
#if defined(FRONT_BFS)
      thrust::device_vector<int>
        current_front(n),
        next_front(n),
        next_front_size(1, 0);
#endif
        thrust::device_vector<float>
        min_norm_sep(n);      // the minimal normalized separator
                              // value, for each component
      thrust::device_vector<bool>
        eliminated(n, false), // bool flags for each node indicating
                              // if the node has been eliminated yet
        running(1);           // bool flag to check whether the BFS
                              // has completed
      thrust::minstd_rand rng;
      auto p = [](auto& v) {
        return thrust::raw_pointer_cast(v.data());
      };
      auto run = [&n](const auto& f) {
        thrust::for_each_n
          (thrust::device, thrust::counting_iterator<int>(0), n, f);
      };
      // int lvl = 0;
      do {
        // std::cout << "lvl: " << lvl++
        //           << " nr_eliminated: " << nr_eliminated[0] << std::endl;
        thrust::sequence(label.begin(), label.end());
        do {
          // find all conn-comps using label prop BFS
          running[0] = false;
          run(bfs_label_prop_ftor<integer>
              (p(ptr), p(ind), p(eliminated), p(label), p(running)));
        } while (running[0]);
        d_nr_comps[0] = 0;
        run(find_comps_ftor
            (p(d_nr_comps), p(eliminated), p(label),
             p(comp), p(comp_root)));
        int nr_comps = d_nr_comps[0];
        thrust::gather // comp[i] = comp[label[i]];
          (label.begin(), label.end(), comp.begin(), comp.begin());
        thrust::fill_n(comp_start.begin(), nr_comps+1, 0);
        run(count_comps_ftor(p(eliminated), nr_comps, p(comp), p(comp_start)));
        thrust::exclusive_scan
          (comp_start.begin(), comp_start.begin()+nr_comps+1,
           comp_start.begin());
        thrust::fill_n(min_norm_sep.begin(), nr_comps,
                       std::numeric_limits<float>::max());

        // TODO eliminate small components with minimum degree

        auto find_level_sets = [&](thrust::device_vector<int>& roots) {
          thrust::fill_n(label.begin(), n, n);
          thrust::scatter // label[roots[i]] = comp_start[i]
            (comp_start.begin(), comp_start.begin()+nr_comps,
             roots.begin(), label.begin());
#if defined(FRONT_BFS)
          thrust::copy_n(roots.begin(), nr_comps, current_front.begin());
          int front_size = nr_comps;
          while (front_size) {
            // BFS from the root node of each component
            next_front_size[0] = 0;
            thrust::for_each_n
              (thrust::device, thrust::counting_iterator<int>(0), front_size,
               bfs_layer_front_ftor<integer>
                (n, p(ptr), p(ind), p(eliminated), p(label),
                 p(current_front), p(next_front), p(next_front_size)));
            std::swap(current_front, next_front);
            front_size = next_front_size[0];
            // repeat until BFS for each component is done
          }
#else
          int it = 0;
          do {
            // BFS from the root node of each component
            running[0] = false;
            run(bfs_layer_ftor<integer>
                (n, it++, p(ptr), p(ind), p(eliminated), p(label),
                 p(comp), p(comp_start), p(running)));
            // repeat until BFS for each component is done
          } while (running[0]);
#endif
          thrust::fill_n(levels.begin(), nr_comps, 1);
          run(count_layers_ftor
              (n, p(eliminated), p(label), p(comp),
               p(levels), p(comp_start)));
        };
        for (int rep=0; rep<3; rep++) {
          // find level sets with different random initial nodes
          if (rep > 0) {
            thrust::copy_n(comp.begin(), n, label.begin());
            thrust::sequence(degree.begin(), degree.end());
            thrust::sort_by_key(label.begin(), label.end(), degree.begin());
            thrust::for_each_n
              (thrust::device, thrust::counting_iterator<int>(0), nr_comps,
               random_roots_ftor
               (rng(), p(comp_root), p(comp_start), p(degree)));
          }
          thrust::fill_n(old_levels.begin(), nr_comps, 1);
          do {
            find_level_sets(comp_root);
            thrust::fill_n
              (thrust::make_zip_iterator(degree.begin(), sep_size.begin()),
               n, thrust::make_tuple(0, 0));
            run(cardinalities_ftor<integer>
                (p(ptr), p(ind), p(eliminated), p(label), p(comp),
                 p(levels), p(comp_start), p(degree), p(sep_size)));
            run(minimize_sep_ftor<integer>
                (p(ptr), p(ind), p(eliminated), p(label), p(comp), p(levels),
                 p(comp_start), p(degree), p(sep_size), p(comp_root),
                 p(comp_root_opt), p(sep_level), p(min_norm_sep)));
            if (thrust::all_of
                (thrust::make_zip_iterator
                 (thrust::make_tuple(levels.begin(), old_levels.begin())),
                 thrust::make_zip_iterator
                 (thrust::make_tuple(levels.begin()+nr_comps,
                                     old_levels.begin()+nr_comps)),
                 tuple_less_equal_ftor()))
              break;
            thrust::fill_n(degree.begin(), nr_comps, n);
            run(peripheral_node_1_ftor<integer>
                (p(ptr), p(ind), p(eliminated), p(label), p(comp),
                 p(levels), p(old_levels), p(degree), p(comp_start)));
            run(peripheral_node_2_ftor<integer>
                (p(ptr), p(ind), p(eliminated), p(label), p(comp),
                 p(levels), p(old_levels), p(degree),
                 p(comp_start), p(comp_root)));
            old_levels = levels;
            // repeat level set BFS with new start nodes, so that the
            // start nodes converge to peripheral nodes of the
            // different components
          } while (1);
        }
        find_level_sets(comp_root_opt);
        run(eliminate_sep_ftor<integer>
            (p(ptr), p(ind), p(eliminated), p(label), p(comp),
             p(levels), p(sep_level), p(nr_eliminated), p(perm)));
      } while (nr_eliminated[0] < n);
      thrust::reverse(perm.begin(), perm.end());
    }

    template<typename integer> SeparatorTree<integer>
    nd_bfs_cuda(integer n, integer* ptr, integer* ind,
                std::vector<integer>& perm, std::vector<integer>& iperm) {
      integer nnz = ptr[n];
      thrust::device_vector<integer> dptr(n+1), dind(nnz), dperm(n);
      thrust::copy(ptr, ptr+n+1, dptr.begin());
      thrust::copy(ind, ind+nnz, dind.begin());
      nd_bfs_device(n, dptr, dind, dperm);
      thrust::copy(dperm.begin(), dperm.end(), perm.begin());
      for (integer i=0; i<n; i++)
        iperm[perm[i]] = i;
      return build_sep_tree_from_perm(ptr, ind, iperm, perm);
    }

    template SeparatorTree<int>
    nd_bfs_cuda(int n, int* ptr, int* ind,
                std::vector<int>& perm, std::vector<int>& iperm);
    template SeparatorTree<long int>
    nd_bfs_cuda(long int n, long int* ptr, long int* ind,
                std::vector<long int>& perm, std::vector<long int>& iperm);
    template SeparatorTree<long long int>
    nd_bfs_cuda(long long int n, long long int* ptr, long long int* ind,
                std::vector<long long int>& perm,
                std::vector<long long int>& iperm);

  } // end namespace ordering
} // end namespace strumpack
